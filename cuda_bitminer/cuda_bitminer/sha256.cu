#include "hip/hip_runtime.h"
#pragma once
#include <hip/hip_runtime_api.h>	
#include ""

#include <sstream>
#include "sha256.cuh"


#define N 512
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#define ROTRIGHT(a,b) (((a) >> (b)) | ((a) << (32-(b))))
#define SIGMA0(a) (ROTRIGHT(a,7) ^ ROTRIGHT(a,18) ^ ((a) >> 3))
#define SIGMA1(a) (ROTRIGHT(a,17) ^ ROTRIGHT(a,19) ^ ((a) >> 10))
#define CH(a,b,c) (((a) & (b)) ^ (~(a) & (c)))
#define MAJ(a,b,c) (((a) & (b)) ^ ((a) & (c)) ^ ((b) & (c)))
#define SUM0(a) (ROTRIGHT(a,2) ^ ROTRIGHT(a,13) ^ ROTRIGHT(a,22))
#define SUM1(a) (ROTRIGHT(a,6) ^ ROTRIGHT(a,11) ^ ROTRIGHT(a,25))

using namespace std;


__global__ void addKernel(int* c, int* a, int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

void GPUSHA256(void* gpu3, void* gpu1, void* gpu2) {
	addKernel << < 1, 10 >> > ((int*)gpu3, (int*)gpu1, (int*)gpu2);
}

//__global__ void WORK(WORD* src, WORD* result)
//{
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//	if (idx > 15) {
//		result[idx] = SIGMA1(src[idx - 2]) + src[idx - 7] + SIGMA0(src[idx - 15]) + src[idx - 16];
//	}
//}




string WORDToStr(WORD* h, string text1) {

	// HEX BASARKEN BASTAKI 0 HEXADECIMAL SAYILARI BASMIYORDU, IF ILE ONARILDI !
	// "Bize her yer Trabzon! Bolumun en yakisikli hocasi Ibrahim Hoca'dir.000"
	// 0b208c73de9744cf1aab8788d8217f24a0d34b98c6d0e7c17e16e0c4b84db276
	
	string result;
	for (size_t i = 0; i < 8; i++)
	{
		std::stringstream ss;
		ss << hex << h[i];
		for (size_t j = 0; j < 8 - ss.str().length(); j++)
			result += "0";
		result += ss.str();
	}

	// DEBUG ICIN
	if (result.length() != 64) {
		string result;
		for (size_t i = 0; i < 8; i++)
		{
			std::stringstream ss;
			ss << hex << h[i];
			for (size_t j = ss.str().length(); j < 8; j++)
				result += "0";
			result += ss.str();
		}
	}

	return result;
}

//string decToBinary(uint n)
//{
//	uint i = 0, binaryNum[64];
//
//	while (n > 0) {
//		binaryNum[i] = n % 2;
//		n = n / 2;
//		i++;
//	}
//
//	string result;
//	for (int step = i - 1; step >= 0; step--) {
//		result += to_string(binaryNum[step]);
//	}
//	return result;
//}
//
//void decToHexa(int num)
//{
//	// char array to store hexadecimal number
//	char hexaDeciNum[64];
//	int i = 0, temp = 0;
//
//	while (num != 0) {
//
//		temp = num % 16;
//
//		if (temp < 10) hexaDeciNum[i] = temp + 48;
//		else hexaDeciNum[i] = temp + 55;
//
//		num = num / 16;
//		i++;
//	}
//
//	// hexadecimal number array in reverse order
//	char temp1;
//	int j = 0;
//	for (int step = i - 1; step >= 0; step--) {
//		temp1 = hexaDeciNum[step];
//		hexaDeciNum[step] = hexaDeciNum[j];
//		hexaDeciNum[j] = temp1;
//		j++;
//	}
//}


WORD* InitializeHashValues() {

	WORD* h = new WORD[8];
	h[0] = 0x6a09e667;
	h[1] = 0xbb67ae85;
	h[2] = 0x3c6ef372;
	h[3] = 0xa54ff53a;
	h[4] = 0x510e527f;
	h[5] = 0x9b05688c;
	h[6] = 0x1f83d9ab;
	h[7] = 0x5be0cd19;
	return h;
}

WORD* InitializeKValues() {

	WORD* k = new WORD[64]{
	0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2 };

	return k;
}


void WordExtend(WORD* words) {

	// seri olmak zorunda
	for (size_t i = 16; i < 64; i++)
		words[i] = SIGMA1(words[i - 2]) + words[i - 7] + SIGMA0(words[i - 15]) + words[i - 16];
}

void WordCompress(WORD* abcdefgh, WORD* k, WORD* expandedWords) {

	// seri olmak zorunda, YALNIZCA FOR ICI PARALEL YAPILABILIR, 64 TH OLMAZ
	WORD temp1, temp2;
	for (size_t i = 0; i < 64; ++i) {
		temp1 = abcdefgh[7] + SUM1(abcdefgh[4]) + CH(abcdefgh[4], abcdefgh[5], abcdefgh[6]) + k[i] + expandedWords[i];
		temp2 = SUM0(abcdefgh[0]) + MAJ(abcdefgh[0], abcdefgh[1], abcdefgh[2]);
		abcdefgh[7] = abcdefgh[6];
		abcdefgh[6] = abcdefgh[5];
		abcdefgh[5] = abcdefgh[4];
		abcdefgh[4] = abcdefgh[3] + temp1;
		abcdefgh[3] = abcdefgh[2];
		abcdefgh[2] = abcdefgh[1];
		abcdefgh[1] = abcdefgh[0];
		abcdefgh[0] = temp1 + temp2;
	}
}

WORD* SHA256(WORD* words, WORD* h, WORD* k) {

	WORD* expandedWord = new WORD[64]();
	memcpy(expandedWord, words, 16 * sizeof(WORD));
	// kelime genisletme algoritmasi
	WordExtend(expandedWord);

	// initialize hash values copy to abcdefgh
	WORD* abcdefgh = new WORD[8];	// 8 * 4 = 32 Byte => 256 bits
	memcpy(abcdefgh, h, 8 * sizeof(WORD));

	// word compress algoritmasi
	WordCompress(abcdefgh, k, expandedWord);

	abcdefgh[0] = h[0] + abcdefgh[0];
	abcdefgh[1] = h[1] + abcdefgh[1];
	abcdefgh[2] = h[2] + abcdefgh[2];
	abcdefgh[3] = h[3] + abcdefgh[3];
	abcdefgh[4] = h[4] + abcdefgh[4];
	abcdefgh[5] = h[5] + abcdefgh[5];
	abcdefgh[6] = h[6] + abcdefgh[6];
	abcdefgh[7] = h[7] + abcdefgh[7];

	return abcdefgh;
}

WORD* BinTextToWORD(const char* text) {

	// tum degerler 0 ile initialize edilir
	WORD* words = new WORD[64]();

	// 16 TH ILE PARALLEL 
	// gelen blokta her biri, bir biti temsil eden 16 * 32 => 512 karakter var
	for (size_t i = 0; i < 16; i++)
	{
		for (size_t j = 0; j < 32; j++)
		{
			words[i] = ROTLEFT(words[i], 1);
			if (text[i * 32 + j] == *"1")
				words[i] = words[i] | (WORD)1;
		}
	}

	return words;
}

// PARALLEL
string TextToBinaryStr(string words)
{
	string binaryString = "";
	for (char& _char : words) {
		binaryString += bitset<8>(_char).to_string();
	}
	return binaryString;
}

// PARALLEL
string PreSHA256(string text, size_t& textLen) {

	string binText = TextToBinaryStr(text);
	size_t binTextLen = binText.length();							// text in binary halinin uzunlugu

	string textLenBin = bitset<64>(text.length() * 8).to_string();	// text uzunlugunun binary degeri => 64 bits uzunlugunda

	int padding = N - (binTextLen + textLenBin.length()) % N;

	binText += "1";
	for (int i = 1; i < padding; i++)
		binText += "0";

	binText += textLenBin;
	textLen = binText.length();

	return binText;
}

WORD* RecursiveSHA256(const char* binText, WORD* h, WORD* k, size_t textLen, size_t step) {

	if (step == textLen)
		return h;

	WORD* wordBlock = BinTextToWORD(&binText[step]);
	WORD* hash = SHA256(wordBlock, h, k);
	RecursiveSHA256(binText, hash, k, textLen, step + N);
}